#include "hip/hip_runtime.h"
#include "ratings_cuda.h"
#include "timer.h"

#include <algorithm>

__global__ void ratings_error(int players, double* ratings, double* scores, double* error,
int* indices, int* opp_played, int* opp_index)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= players)
  {
    return;
  }

  double score = 0;
  for (int k = indices[i]; k != indices[i+1]; ++k)
  {
    score += opp_played[k] * ratings[i] / (ratings[i] + ratings[opp_index[k]]);
  }
  error[i] = scores[i] - score;
}

__global__ void adjust_ratings(int players, double K, double* ratings, 
  double* error, int* played)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= players)
  {
    return;
  }

  ratings[i] = ratings[i] * pow(10.0, K * error[i] / played[i]);
}

template <typename T>
void cuda_copy(const std::vector<T>& v, T*& dest)
{
  hipMalloc(&dest, v.size() * sizeof(T)); 
  hipMemcpy(dest, v.data(), v.size()*sizeof(T), hipMemcpyHostToDevice);
}

void RatingsCuda::find_ratings()
{
  cuda_copy(ratings_, d_ratings);
  cuda_copy(scores, d_scores);
  cuda_copy(opp_played, d_opp_played);
  cuda_copy(opp_index, d_opp_index);
  cuda_copy(indices_, d_indices);
  cuda_copy(played_, d_played);

  hipMalloc(&d_errors, players_*sizeof(double));

  const int block_size = 128;

  for (int i = 0; i != 3000; ++i)
  {
    ratings_error<<<(players_+1)/block_size, block_size>>>(players_, d_ratings, d_scores,
      d_errors, d_indices, d_opp_played, d_opp_index);

    adjust_ratings<<<players_+1/block_size, block_size>>>(players_, 1.6, d_ratings, d_errors,
    d_played);
  }

  hipMemcpy(ratings_.data(), d_ratings, players_ * sizeof(double),
    hipMemcpyDeviceToHost);
  hipMemcpy(errors_.data(), d_errors, players_ * sizeof(double),
    hipMemcpyDeviceToHost);

  hipFree(d_ratings);
  hipFree(d_scores);
  hipFree(d_errors);
  hipFree(d_indices);
  hipFree(d_opp_played);
  hipFree(d_opp_index);
  hipFree(d_played);
}
